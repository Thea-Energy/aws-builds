#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <stdlib.h>

// Error checking macro for NVML calls
#define CHECK_NVML(call) \
    do { \
        nvmlReturn_t result = call; \
        if (result != NVML_SUCCESS) { \
            printf("NVML Error: %s at line %d\n", nvmlErrorString(result), __LINE__); \
            return 1; \
        } \
    } while(0)

int main() {
    // Get SLURM_LOCALID
    char* slurm_localid = getenv("SLURM_LOCALID");
    int local_id = slurm_localid ? atoi(slurm_localid) : -1;
    
    // Initialize NVML
    CHECK_NVML(nvmlInit());

    // Get NVML device handle for current device
    nvmlDevice_t device;
    CHECK_NVML(nvmlDeviceGetHandleByIndex(local_id, &device));

    // Get UUID
    char uuid[NVML_DEVICE_UUID_BUFFER_SIZE];
    CHECK_NVML(nvmlDeviceGetUUID(device, uuid, NVML_DEVICE_UUID_BUFFER_SIZE));

    // Print only SLURM_LOCALID and UUID
    printf("SLURM_LOCALID=%s, UUID=%s\n", 
           slurm_localid ? slurm_localid : "not set", 
           uuid);

    // Shutdown NVML
    CHECK_NVML(nvmlShutdown());

    return 0;
}
